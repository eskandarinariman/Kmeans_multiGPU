#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <assert.h>
extern "C" {
#include "kmeans.h"
}

/*
 * data       [nvectors  * ndims]
 * clusters   [nclusters * ndims]
 * membership [nvectors]
 */
__device__ inline void
vector_dist(unsigned int vector, const float *data, const float *clusters,
		int *membership, int ndims, int nclusters)
{
	int index = -1;
	float min_dist = FLT_MAX;

	for (int i = 0; i < nclusters; i++) {
		float dist = 0.0;

		for (int j = 0; j < ndims; j++) {
			float diff = data[vector * ndims + j] - clusters[i * ndims + j];
			dist += diff * diff;
		}

		if (dist < min_dist) {
			min_dist = dist;
			index    = i;
		}
	}
	membership[vector] = index;
}

__global__ void
kmeans_one_vector(const float *data, const float *clusters, int *membership,
		int ndims, int nclusters)
{
	unsigned int vector = blockIdx.x * blockDim.x + threadIdx.x;
	vector_dist(vector, data, clusters, membership, ndims, nclusters);
}

__global__ void
kmeans_max_threads(const float *data, const float *clusters, int *membership,
		int ndims, int nclusters, int nvectors, int thread_vectors)
{
	unsigned int start = (blockIdx.x * blockDim.x + threadIdx.x) * thread_vectors;
	unsigned int end   = start + thread_vectors;
	for (int vector = start; vector < end; vector++) {
		if (vector < nvectors)
			vector_dist(vector, data, clusters, membership, ndims, nclusters);
	}
}

__global__ void
kmeans_coalesce(const float *data, const float *clusters, int *membership,
		int ndims, int nclusters, int nvectors, int threads)
{
	unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
	for (int vector = tid; vector < nvectors; vector+=threads) {
		if (vector < nvectors)
			vector_dist(vector, data, clusters, membership, ndims, nclusters);
	}
}

/*
 * [hd]_data          [nvectors  * ndims]
 * [hd]_clusters      [nclusters * ndims]
 * [hd]_membership    [nvectors]
 * h_clusters_members [nclusters]
 * h_clusters_sums    [nclusters * ndims]
 */
extern "C" int
run_kmeans(const float *h_data, const float *d_data, float *h_clusters,
		float *d_clusters, int *h_membership, int *d_membership,
		int *h_clusters_members, float *h_clusters_sums, long nvectors,
		int ndims, int nclusters, int niters)
{
	//todo: make threadsPerBock and
#ifdef ONE_VECTOR
	int thread_vectors = 1;
	int block_threads = 64;
	assert(nvectors % thread_vectors == 0);
	assert((nvectors / thread_vectors) % block_threads == 0);
	int grid_blocks = (nvectors / thread_vectors) / block_threads;
#elif MAX_THREADS || COALESCE
	int grid_blocks = 128;
	int block_threads = 16;
	int threads = grid_blocks * block_threads;
	assert(threads == 2048);
#if MAX_THREADS
	int thread_vectors = (nvectors + (threads - 1))/threads;
#endif
#endif

	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

//	for (int i = 0; i < niters; i++) {
		hipError_t err = hipMemcpy(d_clusters, h_clusters, nclusters * ndims * sizeof(float),
				hipMemcpyHostToDevice);
		if (err != hipSuccess) {
			fprintf(stderr, "cudamemcpy d_clusters error %s\n", hipGetErrorString(err));
			return -1;
		}

#ifdef ONE_VECTOR
		kmeans_one_vector<<<grid_blocks, block_threads>>>(d_data, d_clusters,
				d_membership, ndims, nclusters);
#elif MAX_THREADS
		kmeans_max_threads<<<grid_blocks, block_threads>>>(d_data, d_clusters,
				d_membership, ndims, nclusters, nvectors, thread_vectors);
#elif COALESCE
		kmeans_coalesce<<<grid_blocks, block_threads>>>(d_data, d_clusters,
				d_membership, ndims, nclusters, nvectors, threads);
#endif
		err = hipGetLastError();
		if (err != hipSuccess) {
			fprintf(stderr, "kmeans_kernel error %s\n", hipGetErrorString(err));
			return -1;
		}
		hipDeviceSynchronize();

		err = hipMemcpy(h_membership, d_membership, nvectors * sizeof(int), hipMemcpyDeviceToHost);
		if (err != hipSuccess) {
			fprintf(stderr, "cudamemcpy h_membership error %s\n", hipGetErrorString(err));
			return -1;
		}

		cpu_sum_clusters(h_data, h_membership, h_clusters_members,
				h_clusters_sums, nvectors, ndims, nclusters);

		// for (int i = 0; i < nclusters; i++)
		// 	for (int j = 0; j < ndims; j++)
		// 		h_clusters[i * ndims + j] = h_clusters_sums[i * ndims + j] / h_clusters_members[i];
//	}

	clock_gettime(CLOCK_MONOTONIC, &end);

	// printf("runtime = %luns\n", time_diff(start, end));
	return 0;
}

/*
 * data               [nvectors  * ndims]
 * membership         [nvectors]
 * h_clusters_members [nclusters]
 * h_clusters_sums    [nclusters * ndims]
 */
void
cpu_sum_clusters(const float *data, const int *membership, int *clusters_members,
		float *clusters_sums, long nvectors, int ndims, int nclusters)
{
	for (int i = 0; i < nvectors; i++) {
		int cluster = membership[i];
		clusters_members[cluster]++;
		for (int j = 0; j < ndims; j++)
			clusters_sums[cluster * ndims + j] += data[i * ndims + j];
	}
}

/*
 * [hd]_data          [nvectors  * ndims]
 * [hd]_clusters      [nclusters * ndims]
 * [hd]_membership    [nvectors]
 * h_clusters_members [nclusters]
 * h_clusters_sums    [nclusters * ndims]
 */
extern "C" int
device_setup_data(float **h_data, float **d_data, float **d_clusters,
		int **d_membership, long nvectors, int ndims, int nclusters)
{

	hipError_t err = hipMalloc(d_data, nvectors * ndims * sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "cudamalloc d_data error %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMemcpy(*d_data, *h_data, nvectors * ndims * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "cudamemcpy d_data error %s\n", hipGetErrorString(err));
		return 1;
	}

	// for (int i = 0; i < nclusters; i++)
	// 	for (int j = 0; j < ndims; j++)
	// 		//(*h_clusters)[i * ndims + j] = (*h_data)[i * ndims + j];
	// 		printf("%f ",(*h_clusters)[i * ndims + j]);

	err = hipMalloc(d_clusters, nclusters * ndims * sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "cudamalloc d_clusters error %s\n", hipGetErrorString(err));
		return -1;
	}

	err = hipMalloc(d_membership, nvectors * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudamalloc d_membership error %s\n", hipGetErrorString(err));
		return -1;
	}

	return 0;
}

extern "C" int
setup_data(float **h_data, float **d_data, float **h_clusters, float **d_clusters,
		int **h_membership, int **d_membership, int **h_clusters_members,
		float **h_clusters_sums, long nvectors, int ndims, int nclusters, const char *infile)
{
	*h_data = (float *)malloc(nvectors * ndims * sizeof(float));
	if (*h_data == NULL) {
		fprintf(stderr, "malloc h_data failed\n");
		return 1;
	}

	int errr = read_data(h_data, nvectors * ndims * sizeof(float), infile);
	if (errr) {
		fprintf(stderr, "read_data error: %d\n", errr);
		return 1;
	}

	hipError_t err = hipMalloc(d_data, nvectors * ndims * sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "cudamalloc d_data error %s\n", hipGetErrorString(err));
		return 1;
	}

	err = hipMemcpy(*d_data, *h_data, nvectors * ndims * sizeof(float), hipMemcpyHostToDevice);
	if (err != hipSuccess) {
		fprintf(stderr, "cudamemcpy d_data error %s\n", hipGetErrorString(err));
		return 1;
	}

	*h_clusters = (float *)malloc(nclusters * ndims * sizeof(float));
	if (*h_clusters == NULL) {
		fprintf(stderr, "malloc h_clusters failed\n");
		return 1;
	}

	for (int i = 0; i < nclusters; i++){
		for (int j = 0; j < ndims; j++){
			(*h_clusters)[i * ndims + j] = (*h_data)[i * ndims + j];
			//printf("%f ",(*h_clusters)[i * ndims + j]);
		}
		//printf("\n");
	}

	err = hipMalloc(d_clusters, nclusters * ndims * sizeof(float));
	if (err != hipSuccess) {
		fprintf(stderr, "cudamalloc d_clusters error %s\n", hipGetErrorString(err));
		return -1;
	}

	*h_membership = (int *)malloc(nvectors * sizeof(int));
	if (*h_membership == NULL) {
		fprintf(stderr, "malloc h_membership failed\n");
		return -1;
	}

	err = hipMalloc(d_membership, nvectors * sizeof(int));
	if (err != hipSuccess) {
		fprintf(stderr, "cudamalloc d_membership error %s\n", hipGetErrorString(err));
		return -1;
	}

	*h_clusters_members = (int *)malloc(nclusters * sizeof(int));
	if (*h_clusters_members == NULL) {
		fprintf(stderr, "malloc h_clusters_members failed\n");
		return -1;
	}

	*h_clusters_sums = (float *)malloc(nclusters * ndims * sizeof(float));
	if (*h_clusters_sums == NULL) {
		fprintf(stderr, "malloc h_clusters_sums failed\n");
		return -1;
	}

	return 0;
}

// int
// main(int argc, char *argv[])
// {
// 	if (argc != 6) {
// 		printf("usage: ./kmeans <infile> <vectors> <dimensions> <clusters> <iterations>\n");
// 		return 1;
// 	}

// 	char *infile  = argv[1];
// 	// need to be careful with large sizes nvectors * ndims can overflow a signed int
// 	long nvectors  = strtol(argv[2], NULL, 10);
// 	int  ndims     = atoi(argv[3]);
// 	int  nclusters = atoi(argv[4]);
// 	int  niters    = atoi(argv[5]);

// 	float *h_data, *d_data, *h_clusters, *d_clusters, *h_clusters_sums;
// 	int *h_membership, *d_membership, *h_clusters_members;
// 	int err = setup_data(&h_data, &d_data, &h_clusters, &d_clusters,
// 			&h_membership, &d_membership, &h_clusters_members,
// 			&h_clusters_sums, nvectors, ndims, nclusters, infile);
// 	if (err)
// 		return err;

// 	printf("setup complete running kmeans...\n");

// 	err = run_kmeans(h_data, d_data, h_clusters, d_clusters, h_membership,
// 			d_membership, h_clusters_members, h_clusters_sums, nvectors, ndims,
// 			nclusters, niters);
// 	if (err)
// 		return err;

// 	free(h_data);
// 	hipFree(d_data);
// 	free(h_clusters);
// 	hipFree(d_clusters);
// 	free(h_membership);
// 	hipFree(d_membership);
// 	free(h_clusters_members);
// 	free(h_clusters_sums);

// 	hipDeviceReset();
// 	return err;
// }
